#include "hip/hip_runtime.h"
#include<iostream>    // cout, endl
#include<algorithm>   // iota, fill
#include<omp.h>       // benchmark below (mutli-threading with openMP pragmas)

///////////////////////////////////////////////////////////////////////////////
// IGNORE THESE HELPERS (taken from https://github.com/gravitino/cudahelpers)
///////////////////////////////////////////////////////////////////////////////

// safe division
#define SDIV(x,y)(((x)+(y)-1)/(y))

// error makro
#define CUERR {                                                              \
    hipError_t err;                                                         \
    if ((err = hipGetLastError()) != hipSuccess) {                         \
       std::cout << "CUDA error: " << hipGetErrorString(err) << " : "       \
                 << __FILE__ << ", line " << __LINE__ << std::endl;          \
       exit(1);                                                              \
    }                                                                        \
}

// convenient timers
#define TIMERSTART(label)                                                    \
        hipEvent_t start##label, stop##label;                               \
        float time##label;                                                   \
        hipEventCreate(&start##label);                                      \
        hipEventCreate(&stop##label);                                       \
        hipEventRecord(start##label, 0);

#define TIMERSTOP(label)                                                     \
        hipEventRecord(stop##label, 0);                                     \
        hipEventSynchronize(stop##label);                                   \
        hipEventElapsedTime(&time##label, start##label, stop##label);       \
        std::cout << "#" << time##label                                      \
                  << " ms (" << #label << ")" << std::endl; 


///////////////////////////////////////////////////////////////////////////////
// STUDENTS' PART (feel free to code)
// compile with: nvcc vector_add.cu -std=c++11 -O3 -arch=sm_30 \
//               -Xcompiler="-fopenmp" -o vector_add
///////////////////////////////////////////////////////////////////////////////

// 1 gigabytes of memory for float (device 0 has 12G RAM in total)
#define N (1L<<28)

__global__
void add_kernel(float * A, float * B, float * C, size_t n) {
    int thid = blockDim.x*blockIdx.x+threadIdx.x;

    if (thid < n)
        C[thid] = A[thid]+B[thid];
}


__global__
void sgr_kernel(float * A, float * B, float * C, size_t n) {
    int thid = blockDim.x*blockIdx.x+threadIdx.x;

    for (int index = thid; index < n; index += gridDim.x*blockDim.x)
         C[index] = A[index]+B[index];
}

int main () {
    
    // use the first GPU (0..Tesla K40 12G RAM, 1..Titan 6G RAM)
    hipSetDevice(0);                                                     CUERR

    // small letters for host, capital letters for device memory
    float *a, *b, *c, *A, *B, *C;

    // allocate host memory
    hipHostMalloc(&a, sizeof(float)*N);                                  CUERR
    hipHostMalloc(&b, sizeof(float)*N);                                  CUERR
    hipHostMalloc(&c, sizeof(float)*N);                                  CUERR
    
    // allocate device memory
    hipMalloc(&A, sizeof(float)*N);                                      CUERR
    hipMalloc(&B, sizeof(float)*N);                                      CUERR
    hipMalloc(&C, sizeof(float)*N);                                      CUERR

    // fill a and b with stuff
    TIMERSTART(fillArraysOnHostSide)
    std::iota(a, a+N, 0);             // (0, 1, 2, 3, ..., N-1)
    std::fill(b, b+N, 1);             // (1, 1, 1, 1, ..., 1)
    TIMERSTOP(fillArraysOnHostSide)

    // measure the time for overall execution on GPU
    TIMERSTART(overallCUDA)

    // copy a and b to A and B from host to device
    TIMERSTART(copyHostToDevice)
    hipMemcpy(A, a, sizeof(float)*N, hipMemcpyHostToDevice);            CUERR
    hipMemcpy(B, b, sizeof(float)*N, hipMemcpyHostToDevice);            CUERR
    TIMERSTOP(copyHostToDevice)

    // Note, the next line is not needed in practice. However, we overwrite
    // the device vector C to prevent spurious false positives. As an example,
    // if another student writes the correct result to C and the GPU assigns
    // the same address range during your run (this happens quite often) then
    // you might pass the test below even if you process nothing!
    hipMemset(C, 0, sizeof(float)*N);                                    CUERR

    // invoke the kernel
    TIMERSTART(kernelTime)
    add_kernel<<<SDIV(N, 1024), 1024>>>(A, B, C, N);                      CUERR
    TIMERSTOP(kernelTime)

    // if you are bored try to write the kernel for this grid configuration
    TIMERSTART(staticGridKernelTime)
    sgr_kernel<<<1024, 1024>>>(A, B, C, N);                               CUERR
    TIMERSTOP(staticGridKernelTime)

    // copy C to c from device to host
    TIMERSTART(copyDeviceToHost)
    hipMemcpy(c, C, sizeof(float)*N, hipMemcpyDeviceToHost);            CUERR
    TIMERSTOP(copyDeviceToHost)

    // stop overall GPU timer and print result
    TIMERSTOP(overallCUDA)

    ///////////////////////////////////////////////////////////////////////////
    // BENCHMARKS AND CHECKS (you may ignore this, especially the openMP part)
    ///////////////////////////////////////////////////////////////////////////

    // check for correct result computed by CUDA
    for (size_t index = 0; index < N; index++) {
        if (c[index] != a[index]+b[index]) {
            std::cout << "error at position " << index << std::endl;
            break;
        }
    }

    // measure time for vector addition on single-threaded host
    TIMERSTART(overallSingleCore)
    for (size_t index = 0; index < N; index++)
        c[index] = a[index]+b[index];
    TIMERSTOP(overallSingleCore)

    // measure time for vector addition on multi-threaded host
    TIMERSTART(overallMultiCore)
    # pragma omp parallel for
    for (size_t index = 0; index < N; index++)
        c[index] = a[index]+b[index];
    TIMERSTOP(overallMultiCore)

    // get rid of the memory
    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);

    // print status
    float usedMem = 3.0*N*sizeof(float)/(1L<<30);
    std::cout << "#processed " << usedMem << " gigabytes." << std::endl;
    std::cout << "CUDA programming is fun!" << std::endl;
}
